/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
	const torch::Tensor& ref_colors,//new
    const torch::Tensor& opacity,
	const torch::Tensor& ref_opacity,//new
    const torch::Tensor& beta,//new
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const torch::Tensor& ref_sh,//new
	const int degree,
	const int ref_degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool antialiasing,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor trans_weights = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor comp_ref_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor comp_trans_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  
  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor out_invdepth = torch::full({0, H, W}, 0.0, float_opts);
  float* out_invdepthptr = nullptr;

  out_invdepth = torch::full({1, H, W}, 0.0, float_opts).contiguous();
  out_invdepthptr = out_invdepth.data<float>();

  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }
	  int ref_M = 0;
	  if(ref_sh.size(0) != 0)
	  {
		ref_M = ref_sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, ref_degree, M, ref_M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		ref_sh.contiguous().data_ptr<float>(),//new
		ref_colors.contiguous().data<float>(), //new
		opacity.contiguous().data<float>(), 
		ref_opacity.contiguous().data<float>(), //new
		beta.contiguous().data<float>(), //new
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		trans_weights.contiguous().data<float>(),//new
		comp_ref_color.contiguous().data<float>(),//new
		comp_trans_color.contiguous().data<float>(),//new
		out_color.contiguous().data<float>(),
		out_invdepthptr,
		antialiasing,
		radii.contiguous().data<int>(),
		debug);
  }
//   return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, out_invdepth);
	return std::make_tuple(rendered, trans_weights, comp_ref_color, comp_trans_color, out_color, radii, geomBuffer, binningBuffer, imgBuffer, out_invdepth);

}

// std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
  RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& ref_colors,//new
    const torch::Tensor& betas,//new
	const torch::Tensor& opacities,
	const torch::Tensor& ref_opacities,//new
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
	const torch::Tensor& dL_dref_map,//new
    const torch::Tensor& dL_dref_color,//new
    const torch::Tensor& dL_dtrans_color,//new
    // const torch::Tensor& dL_dout_color,
	const torch::Tensor& dL_dout_invdepth,
	const torch::Tensor& sh,
	const torch::Tensor& ref_sh,//new
	const int degree,
	const int ref_degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const torch::Tensor& trans_weights,//new
	const torch::Tensor& comp_ref_color,//new
	const bool antialiasing,
	const bool debug)
{
  const int P = means3D.size(0);
  const int H = dL_dref_color.size(1);
  const int W = dL_dref_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }
  int ref_M = 0;
  if(ref_sh.size(0) != 0)
  {	
	ref_M = ref_sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dtranscolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_drefcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_drefopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dbeta = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_drefsh = torch::zeros({P, ref_M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dinvdepths = torch::zeros({0, 1}, means3D.options());
  
  float* dL_dinvdepthsptr = nullptr;
  float* dL_dout_invdepthptr = nullptr;
  if(dL_dout_invdepth.size(0) != 0)
  {
	dL_dinvdepths = torch::zeros({P, 1}, means3D.options());
	dL_dinvdepths = dL_dinvdepths.contiguous();
	dL_dinvdepthsptr = dL_dinvdepths.data<float>();
	dL_dout_invdepthptr = dL_dout_invdepth.data<float>();
  }

  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, ref_degree, M, ref_M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  ref_sh.contiguous().data<float>(),//new
	  ref_colors.contiguous().data<float>(),//new
	  betas.contiguous().data<float>(),//new
	  opacities.contiguous().data<float>(),
	  ref_opacities.contiguous().data<float>(),//new
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  trans_weights.contiguous().data<float>(),//new 
	//   dL_dout_color.contiguous().data<float>(),
	  comp_ref_color.contiguous().data<float>(),//new
	  dL_dref_map.contiguous().data<float>(),//new
	  dL_dref_color.contiguous().data<float>(),//new
	  dL_dtrans_color.contiguous().data<float>(),//new
	  dL_dout_invdepthptr,
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_drefopacity.contiguous().data<float>(),//new
	  dL_dbeta.contiguous().data<float>(),//new
	//   dL_dcolors.contiguous().data<float>(),
	  dL_dtranscolors.contiguous().data<float>(),//new
	  dL_drefcolors.contiguous().data<float>(),//new
	  dL_dinvdepthsptr,
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_drefsh.contiguous().data<float>(),//new
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  antialiasing,
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dtranscolors, dL_drefcolors, dL_dopacity, dL_drefopacity, dL_dbeta, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_drefsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}
